#include "hip/hip_runtime.h"
#include <stdio.h>
#include "bingo_card_sim_data.h"

#define MAX_NUMBER_CALL 32

#define AlignByteSizeBy(TotalBytes, AlignLength) ((TotalBytes) += ((TotalBytes) % (AlignLength)))

//NOTE(gerald): Reference from "https://en.wikipedia.org/wiki/Xorshift"
__device__ u32
XorShift32(u32 State)
{
    u32 Result = State;
    Result ^= Result << 13;
    Result ^= Result >> 17;
    Result ^= Result << 5;
//    State = Result;
    return Result;
}


__global__ void
LoadNumbersToBeCalled(numbers_load_memory_block GlobalData)
{
    extern __shared__ u8 SliceData[];

    u32 threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
    u32 numberIndex = threadID % MAX_NUMBER_CALL;
    u32 SliceIndex = threadIdx.x / MAX_NUMBER_CALL;
    if(numberIndex == 0)
    {
        //StartNumber for this slice
        SliceData[SliceIndex * 2] = 1 + (XorShift32(threadID+1) % GlobalData.MaxNumbers);
        //IncrementNumber for this slice
        SliceData[(SliceIndex*2) + 1] = 7;//7 + (XorShift32(threadID+1) % 15);
    }
    
    __syncthreads();

    GlobalData.NumbersToBeCalled[threadID] =
        1 + (SliceData[SliceIndex * 2] + ((numberIndex+1) * SliceData[(SliceIndex*2) + 1])) % GlobalData.MaxNumbers;
//    printf("ThreadID : %d, No : %d\n", threadIdx.x, GlobalData.NumbersToBeCalled[threadID]);
}

__global__ void
LoadCardDataKernel(card_load_memory_block GlobalData)
{
    __shared__ u8 Incrementer;
    extern __shared__ u8 ColumnSliceData[];

    u32 CellIndex = threadIdx.x % GlobalData.CardStride;
    u32 SliceIndex = threadIdx.x / GlobalData.CardStride;
    u32 threadID = (blockIdx.x * blockDim.x) + threadIdx.x;

    if(threadIdx.x == 0)
    {
        Incrementer = (GlobalData.MaxNumbers == 75) ? 3 : 5;
    }
    
    if(CellIndex == 0)
    {
        for(u32 ColumnIndex = 0;
            ColumnIndex < GlobalData.Row;
            ColumnIndex++)
        {
            u32 InitNumber = 1 + ColumnIndex * GlobalData.MaxNumberPerColumn;
            //StartNumber for this column slice
            ColumnSliceData[(SliceIndex * 2 * GlobalData.Row) + (ColumnIndex * 2)] = InitNumber + (XorShift32(threadID+1+ColumnIndex) % GlobalData.MaxNumberPerColumn);
            //IncrementNumber for this column slice
            ColumnSliceData[(SliceIndex * 2 * GlobalData.Row) + (ColumnIndex * 2) + 1] = InitNumber;
        }
    }
    
    __syncthreads();

    u32 ColumnIndex = CellIndex / GlobalData.Row;
    u32 numberIndex = CellIndex % GlobalData.Row;
    
    GlobalData.Cards[threadID] =
        ColumnSliceData[(SliceIndex * 2 * GlobalData.Row) + (ColumnIndex * 2) + 1] +
        (ColumnSliceData[(SliceIndex * 2 * GlobalData.Row) + (ColumnIndex * 2)] +
                     ((numberIndex+1) * Incrementer)) % GlobalData.MaxNumberPerColumn;
}

__global__ void
CardDaubKernel(memory_block GlobalData)
{
    //TODO(gerald): make sure the stride is aligned nicely
    __shared__ u32 CardStride;
    __shared__ u32 BlockStride;
    __shared__ u8 NumbersToBeCalled[MAX_NUMBER_CALL];//MaxNumbersToBeCalled

    //TODO(gerald): make sure the cards are aligned in a way there are no memory-bank conflicts
    extern __shared__ u32 CardInfos[];//count = sizeof(MaxCardsPerRoom * CardStride * sizeof(u32))
    
    //fill shared block info based on block idx
    if(threadIdx.x == 0)
    {
        CardStride = GlobalData.Row * GlobalData.Column;
        BlockStride = blockIdx.x * blockDim.x;
//        printf("BlockIdx : %d, Stride : %d\n", blockIdx.x, BlockStride);
    }

    __syncthreads();

    //load numbers to be called into shared memory
    if(threadIdx.x < 32)
    {
        NumbersToBeCalled[threadIdx.x] =
            GlobalData.NumbersToBeCalled[(blockIdx.x * MAX_NUMBER_CALL) + threadIdx.x];
    }
    

    __syncthreads();

    //load each card data
    for(u32 CellIndex = 0;
        CellIndex < CardStride;
        CellIndex++)
    {
        CardInfos[(threadIdx.x * CardStride) + CellIndex] =
            GlobalData.Cards[(BlockStride *  CardStride) + (threadIdx.x * CardStride) + CellIndex];
    }

    //daub numbers
    for(u32 DaubIndex = 0;
        DaubIndex < MAX_NUMBER_CALL;
        DaubIndex++)
    {
        for(u32 CellIndex = 0;
            CellIndex < CardStride;
            CellIndex++)
        {

            if((CardInfos[(threadIdx.x * CardStride) + CellIndex] & 0XFF) ==
               NumbersToBeCalled[DaubIndex])
            {
                CardInfos[(threadIdx.x * CardStride) + CellIndex] |= ((DaubIndex+1) << 8); 
                break;
            }
        }
    }

    //write back the results to the global memory
    for(u32 CellIndex = 0;
        CellIndex < CardStride;
        CellIndex++)
    {
        GlobalData.Cards[(BlockStride *  CardStride) + (threadIdx.x * CardStride) + CellIndex] =
            CardInfos[(threadIdx.x * CardStride) + CellIndex];
    }
}

int
main(void)
{
    u32 MaxRow = 5;
    u32 MaxColumn = 5;
    u32 MaxRooms = 1000;
    u32 MaxCardsPerRoom = 256;
    u32 MaxNumbers = 75;//60;

    u32 MaxBlocks = MaxRooms;
    u32 MaxThreadsPerBlock = MaxCardsPerRoom;
    u32 TotalCards = MaxRooms * MaxCardsPerRoom;

    //TODO(gerald):GetDevice list and get the max core based on the devices
    u32 DeviceMaxThreadPerBlock = 1024;
    
    memory_block HostData;
    HostData.Row = MaxRow;
    HostData.Column = MaxColumn;
    HostData.MaxCardCount = MaxThreadsPerBlock;
    HostData.MaxNumbersToBeCalled = MAX_NUMBER_CALL;

    u32 CardStride = HostData.Row * HostData.Column;
    memory_index NumbersCalledSizeInBytes = MaxBlocks * HostData.MaxNumbersToBeCalled * sizeof(u8);
    memory_index CardDataSizeInBytes = MaxBlocks * MaxThreadsPerBlock * CardStride * sizeof(u32);

    memory_index TotalSizeInBytes = NumbersCalledSizeInBytes + CardDataSizeInBytes;
    AlignByteSizeBy(TotalSizeInBytes, 8);

    //allocate & fill host data
    u8* HostMemoryPtr = (u8*)malloc(TotalSizeInBytes);
    HostData.NumbersToBeCalled = (u8*)HostMemoryPtr;
    HostData.Cards = (u32*)(HostMemoryPtr + NumbersCalledSizeInBytes);

//    u8 ColumnMaxNumbers[5] = {6, 6, 6, 7, 7};
    u8 MaxNumberPerColumn = MaxNumbers/HostData.Row;

#if 0    
    //host load numbers to be called
    for(u32 Index = 0;
        Index < HostData.MaxNumbersToBeCalled;
        Index++)
    {
        HostData.NumbersToBeCalled[Index] = 0xFF;
    }

    //host load card data
    for(u32 CellIndex = 0;
        CellIndex < CardStride;
        CellIndex++)
    {
        HostData.Cards[CellIndex] = 0xFF;
    }
#endif
    
    u8* DeviceMemoryPtr = 0;
    hipMalloc(&DeviceMemoryPtr, TotalSizeInBytes);
//    hipMemcpy(DeviceMemoryPtr, HostMemoryPtr, TotalSizeInBytes, hipMemcpyHostToDevice);

    //gpu memory allocations
    memory_block DeviceData;
    DeviceData.Row = HostData.Row;
    DeviceData.Column = HostData.Column;
    DeviceData.MaxCardCount = HostData.MaxCardCount;
    DeviceData.MaxNumbersToBeCalled = MAX_NUMBER_CALL;
    DeviceData.NumbersToBeCalled = (u8*)DeviceMemoryPtr;
    DeviceData.Cards = (u32*)(DeviceMemoryPtr + NumbersCalledSizeInBytes);

    kernel_config NumbersLoadKernel;
    //NOTE(gerald): 256*125 = 32,000threads i.e 32 Numbers load for 1000 rooms
    NumbersLoadKernel.MaxBlocks = 125;
    NumbersLoadKernel.MaxThreadsPerBlock = 256;
    NumbersLoadKernel.DynamicSharedMemorySizeInBytes = 2 * sizeof(u8) * (NumbersLoadKernel.MaxThreadsPerBlock/MAX_NUMBER_CALL);
    AlignByteSizeBy(NumbersLoadKernel.DynamicSharedMemorySizeInBytes, 8);
    
    numbers_load_memory_block NumbersLoadData;
    NumbersLoadData.MaxNumbers = MaxNumbers;
    NumbersLoadData.NumbersToBeCalled = DeviceData.NumbersToBeCalled;
    //gpu load numbers to be called
    LoadNumbersToBeCalled<<<
            NumbersLoadKernel.MaxBlocks,
            NumbersLoadKernel.MaxThreadsPerBlock,
            NumbersLoadKernel.DynamicSharedMemorySizeInBytes
            >>>(NumbersLoadData);

    u32 CardCountPerBlock = DeviceMaxThreadPerBlock/CardStride;
    kernel_config CardLoadKernel;
    CardLoadKernel.MaxThreadsPerBlock = CardCountPerBlock * CardStride;
    CardLoadKernel.MaxBlocks = TotalCards/CardCountPerBlock;
    CardLoadKernel.DynamicSharedMemorySizeInBytes = 2 * sizeof(u8) * DeviceData.Row * CardCountPerBlock;
    AlignByteSizeBy(CardLoadKernel.DynamicSharedMemorySizeInBytes, 8);
    
    card_load_memory_block CardLoadData;
    CardLoadData.Row = DeviceData.Row;
    CardLoadData.Column = DeviceData.Column;
    CardLoadData.CardStride = CardStride;
    CardLoadData.MaxNumbers = MaxNumbers;
    CardLoadData.MaxNumberPerColumn = MaxNumbers/DeviceData.Column;
    CardLoadData.Cards = DeviceData.Cards;

    //gpu load card data
    LoadCardDataKernel<<<
            CardLoadKernel.MaxBlocks,
            CardLoadKernel.MaxThreadsPerBlock,
            CardLoadKernel.DynamicSharedMemorySizeInBytes
            >>>(CardLoadData);

    kernel_config DaubKernel;
    DaubKernel.MaxBlocks = MaxBlocks;
    DaubKernel.MaxThreadsPerBlock = MaxThreadsPerBlock;
    DaubKernel.DynamicSharedMemorySizeInBytes = DeviceData.MaxCardCount * CardStride * sizeof(u32);
    AlignByteSizeBy(DaubKernel.DynamicSharedMemorySizeInBytes, 8);
    //gpu card daub
    CardDaubKernel<<<
        DaubKernel.MaxBlocks,
        DaubKernel.MaxThreadsPerBlock,
        DaubKernel.DynamicSharedMemorySizeInBytes
            >>>(DeviceData);

#if 1    
    hipDeviceSynchronize();
    hipError_t ErrorCode = hipPeekAtLastError();//hipGetLastError();
    const char* ErrorText = hipGetErrorString(ErrorCode);
    printf("%s\n\n", ErrorText);
#endif
    
    //copy device data back to host data
    hipMemcpy(HostMemoryPtr, DeviceMemoryPtr, TotalSizeInBytes, hipMemcpyDeviceToHost);

#if 1
    srand(time(0));
    u32 RoomIndex = rand() % MaxRooms;
    u32 CardIndex = rand() % CardCountPerBlock;
    printf("RoomConfig:\n");
    printf("\tTotalRooms : %d\n", MaxRooms);
    printf("\tCardsPerRoom : %d\n", MaxCardsPerRoom);
    printf("\tMaxNumbers : %d\n", MaxNumbers);
    printf("\n");
    
    printf("RoomIndex : %d\n", RoomIndex);
    printf("%d Nos called in the room :\n", MAX_NUMBER_CALL);
    //print called nos
    for(u32 NumberIndex = 0;
        NumberIndex < MAX_NUMBER_CALL;
        NumberIndex++)
    {
        u32 Index = (RoomIndex * MAX_NUMBER_CALL) + NumberIndex; 
        printf("%d, ", HostData.NumbersToBeCalled[Index]);
    }
    printf("\n\n");

    printf("CardIndex : %d\n", CardIndex);
    printf("CardData : No(DaubedIdx)\n");
    //print card data
    for(u32 ColumnIndex = 0;
        ColumnIndex < MaxColumn;
        ColumnIndex++)

    {
        for(u32 RowIndex = 0;
            RowIndex < MaxRow;
            RowIndex++)
        {
            u32 CellIndex = (RowIndex * MaxColumn) + ColumnIndex;
            u32 Index = (RoomIndex * MaxThreadsPerBlock * CardStride) + (CardIndex * CardStride) + CellIndex;
            u32 CardNo = HostData.Cards[Index] & 0xFF;;
            u32 DaubIndex = (HostData.Cards[Index] >> 8) & 0xFF;
            printf("%2d(%2d)  ", CardNo, DaubIndex);
        }
        printf("\n");
    }
#endif    

    hipFree(DeviceMemoryPtr);
    free(HostMemoryPtr);
    return(0);
}
